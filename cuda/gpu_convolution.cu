#include "hip/hip_runtime.h"
#include <lcutil.h>

#define FILTER(v,x,y) v*filter[-y*3-x+4]

__global__ void cuda_conv(unsigned int width, unsigned int height, unsigned short *filter, unsigned int total, unsigned char *src, unsigned char *dest) {
    // find current thread x and y
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        unsigned int offset = x + y * width;
        unsigned int res = FILTER(src[offset], 0, 0);
        unsigned int left = -1, right = 1, up = -width, down = width;
        // if we are on the left edge, offset to go left should be 0 as we can't go left
        if (x == 0)
            left = 0;
        // similar for other directions
        else if (x == width - 1)
            right = 0;
        if (y == 0)
            up = 0;
        else if (y == height - 1)
            down = 0;
        // multiply neighbouring pixels by their coefficients and add to result
        res += FILTER(src[offset+right+down], -1, -1);
        res += FILTER(src[offset+down], 0, -1);
        res += FILTER(src[offset+left+down], 1, -1);
        res += FILTER(src[offset+right], -1, 0);
        res += FILTER(src[offset+left], 1, 0);
        res += FILTER(src[offset+right+up], -1, 1);
        res += FILTER(src[offset+up], 0, 1);
        res += FILTER(src[offset+left+up], 1, 1);
        // divide result by the sum of all coefficients and write to target
        dest[offset] = (unsigned char)((res / total) & 0xFF);
    }
}

__global__ void cuda_conv_rgb(unsigned int width, unsigned int height, unsigned short *filter, unsigned int total, unsigned char *src, unsigned char *dest) {
    unsigned int x = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < width && y < height) {
        // same as above, but calculate each channel separately
        unsigned int offset = (x + y * width) * 3;
        unsigned int left = -3, right = 3, up = -3*width, down = 3*width;
        if (x == 0)
            left = 0;
        else if (x == width - 1)
            right = 0;
        if (y == 0)
            up = 0;
        else if (y == height - 1)
            down = 0;

        for (int i = 0; i < 3; i++) {
            unsigned int res = FILTER(src[offset], 0, 0);
            res += FILTER(src[offset+right+down], -1, -1);
            res += FILTER(src[offset+down], 0, -1);
            res += FILTER(src[offset+left+down], 1, -1);
            res += FILTER(src[offset+right], -1, 0);
            res += FILTER(src[offset+left], 1, 0);
            res += FILTER(src[offset+right+up], -1, 1);
            res += FILTER(src[offset+up], 0, 1);
            res += FILTER(src[offset+left+up], 1, 1);
            dest[offset] = (unsigned char)((res / total) & 0xFF);

            offset++;
        }
    }
}

__global__ void cuda_memcmp(unsigned char *buf1, unsigned char *buf2, unsigned int len, char *res) {
    // if we find any difference, write 0 to output and return
    for (unsigned int i = 0; i < len; i++)
        if (buf1[i] != buf2[i]) {
            *res = 0;
            return;
        }
    // otherwise write 1
    *res = 1;
}

extern "C" float convolutionGPU(unsigned int width, unsigned int height, unsigned short *filter, unsigned int is_rgb, unsigned int rounds, unsigned int bufsize, unsigned char *buffer){
    unsigned char *dev_buf1, *dev_buf2;
    unsigned short *dev_filter;
    // allocate needed device buffers
    CUDA_SAFE_CALL( hipMalloc((void**)&dev_buf1, bufsize) );
    CUDA_SAFE_CALL( hipMalloc((void**)&dev_buf2, bufsize) );
    CUDA_SAFE_CALL( hipMalloc((void**)&dev_filter, 9*sizeof(unsigned short)) );

    // copy data to device memory
    CUDA_SAFE_CALL( hipMemcpy(dev_buf1, buffer, bufsize, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(dev_filter, filter, 9*sizeof(unsigned short), hipMemcpyHostToDevice) );

    dim3 dimBl(24, 32);
    dim3 dimGr(FRACTION_CEILING(width, 24), FRACTION_CEILING(height, 32));
    unsigned int total = 0;
    for (int i = 0; i < 9; i++)
        total += filter[i];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    char converge = 0;
    char *dev_converge;
    // allocate one byte on the device as a flag for convergence
    CUDA_SAFE_CALL( hipMalloc(&dev_converge, 1) );
    // start with convergence = 0
    CUDA_SAFE_CALL( hipMemcpy(dev_converge, &converge, 1, hipMemcpyHostToDevice) );
    if (is_rgb == 0) {
        // run the convolution function twice, once with buf1 as the source
        // and buf2 as the target and once in reverse
        for (int i = 0; i < rounds / 2; i++) {
            cuda_conv<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf1, dev_buf2);
            cuda_conv<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf2, dev_buf1);
            if ((i & 15) == 15) {
                // every 16 loops (32 rounds), compare the two buffers
                cuda_memcmp<<<1, 1>>>(dev_buf1, dev_buf2, bufsize, dev_converge);
                // copy the flag from the device to the host
                CUDA_SAFE_CALL( hipMemcpy(&converge, dev_converge, 1, hipMemcpyDeviceToHost) );
                // if no change between them, break
                if (converge == 1)
                    break;
            }
        }
        // if the last bit is 1, we need to run it one more time and move the result to dev_buf1
        if (converge == 0 && (rounds & 1) == 1) {
            cuda_conv<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf1, dev_buf2);
            unsigned char *tmp = dev_buf1;
            dev_buf1 = dev_buf2;
            dev_buf2 = tmp;
        }
    } else {
        // same as above but for RGB
        for (int i = 0; i < rounds / 2; i++) {
            cuda_conv_rgb<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf1, dev_buf2);
            cuda_conv_rgb<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf2, dev_buf1);
            if ((i & 15) == 15) {
                cuda_memcmp<<<1, 1>>>(dev_buf1, dev_buf2, bufsize, dev_converge);
                CUDA_SAFE_CALL( hipMemcpy(&converge, dev_converge, 1, hipMemcpyDeviceToHost) );
                if (converge == 1)
                    break;
            }
        }
        if (converge == 0 && (rounds & 1) == 1) {
            cuda_conv_rgb<<<dimGr, dimBl>>>(width, height, dev_filter, total, dev_buf1, dev_buf2);
            unsigned char *tmp = dev_buf1;
            dev_buf1 = dev_buf2;
            dev_buf2 = tmp;
        }
    }
    CUDA_SAFE_CALL( hipFree(dev_converge) );

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    CUDA_SAFE_CALL( hipGetLastError() );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    // copy results back to host memory
    CUDA_SAFE_CALL( hipMemcpy(buffer, dev_buf1, bufsize, hipMemcpyDeviceToHost) );

    CUDA_SAFE_CALL( hipFree(dev_buf1) );
    CUDA_SAFE_CALL( hipFree(dev_buf2) );
    CUDA_SAFE_CALL( hipFree(dev_filter) );
    return milliseconds;
}

